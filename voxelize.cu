#include "hip/hip_runtime.h"
#include <Eigen/Dense>
#include "voxelize.h"

#include <iostream>
#include <fstream>
#include <string>
#include <algorithm>
#include <vector>
#include <cxxopts.hpp>
#include <hip/hip_runtime_api.h>

#ifdef _OPENMP
#include <omp.h>
#endif

#define BLOCK 512
#define MAXLENGTH 256

using namespace Eigen;
using namespace std;

bool verbose_output = false, parallel = false;
int resolution = 16;	
float voxelsize = 100.0f;

Vector3f min_corner, max_corner;

vector<tetrahedra> mesh;
vector<Eigen::Vector3f> vertex;
vector<float> sample_coord_x, sample_coord_y, sample_coord_z;
bool voxels[MAXLENGTH][MAXLENGTH][MAXLENGTH] = {0};
int MAXSIZE = MAXLENGTH * MAXLENGTH * MAXLENGTH;

__device__
int GetIndex(int i, int j, int k)
{
	return (i * MAXLENGTH + j ) * MAXLENGTH + k;
}

inline float min(float a, float b, float c, float d)
{
	return min(min(min(a,b),c),d);
}

inline float max(float a, float b, float c, float d)
{
	return max(max(max(a,b),c),d);
}

void NodesInput(const char *fname)
{
	int input_count, id, polyface = 3, type1 = 0, type2 = 0;
	float x_coord, y_coord, z_coord;

	ifstream fin(fname);
	fin >> input_count >> polyface >> type1 >> type2;

	assert(polyface == 3 && type1 == 0 && type2 == 0);
	vertex.resize(input_count);

	for(int i = 1; i <= input_count; ++i)
	{
		fin >> id >> x_coord >> y_coord >> z_coord;
		assert(id == i);
		vertex[i] << x_coord, y_coord, z_coord;
	}

	if(verbose_output)
		cout << "NodeCount: " << id << endl;
	string message;
	fin.ignore();
	getline(fin, message);
	if(verbose_output)
		cout << message << endl;
}

void ElementInput(const char *fname)
{
	int input_count, id;
	int poly_face = 4, type = 0, id_A, id_B, id_C, id_D;
	
	ifstream fin(fname);
	fin >> input_count >> poly_face >> type;
	
	assert(poly_face == 4 && type == 0);
	mesh.resize(input_count + 1);

	for(int i = 1; i <= input_count; ++i)
	{
		fin >> id >> id_A >> id_B >> id_C >> id_D;
		assert(id == i);
		tetrahedra t;
		t.a = vertex[id_A];
		t.b = vertex[id_B];
		t.c = vertex[id_C];
		t.d = vertex[id_D];
	
		t.bound_min << min(t.a(0), t.b(0), t.c(0), t.d(0)), min(t.a(1), t.b(1), t.c(1), t.d(1)), min(t.a(2), t.b(2), t.c(2), t.d(2));
		t.bound_max << max(t.a(0), t.b(0), t.c(0), t.d(0)), max(t.a(1), t.b(1), t.c(1), t.d(1)), max(t.a(2), t.b(2), t.c(2), t.d(2));
	
		mesh[i] = t;
	}

	if(verbose_output)
		cout << "ElementCount: " << id << endl;
	string message;
	fin.ignore();
	getline(fin, message);
	if(verbose_output)
		cout << message << endl;
}

void MshInput(string fname = "")
{
	string nd = "$Nodes", ednd = "$EndNodes", ele = "$Elements", edele = "$EndElements";
	string s;
	int input_count;
	int id;
	float x_coord, y_coord, z_coord;
	int poly_face = 4, type = 0, id_A, id_B, id_C, id_D;

	if(fname.empty())
	{
		while(cin >> s)
		{
			if(s == nd)
			{
				cin >> input_count;
				vertex.resize(input_count);
				for(int i = 1; i <= input_count; ++i)
				{
					cin >> id >> x_coord >> y_coord >> z_coord;
					assert(id == i);
					vertex[i] << x_coord, y_coord, z_coord;
				}

				if(verbose_output)
					cout << "NodeCount: " << id << endl;
				cin >> s;
				assert(s == ednd);
			}
			if(s == ele)
			{
				cin >> input_count;
				mesh.resize(input_count);
				for(int i = 1; i <= input_count; ++i)
				{
					cin >> id >> poly_face >> type >> id_A >> id_B >> id_C >> id_D;
					assert(id == i && poly_face == 4 && type == 0);
					tetrahedra t;
					t.a = vertex[id_A];
					t.b = vertex[id_B];
					t.c = vertex[id_C];
					t.d = vertex[id_D];
				
					t.bound_min << min(t.a(0), t.b(0), t.c(0), t.d(0)), min(t.a(1), t.b(1), t.c(1), t.d(1)), min(t.a(2), t.b(2), t.c(2), t.d(2));
					t.bound_max << max(t.a(0), t.b(0), t.c(0), t.d(0)), max(t.a(1), t.b(1), t.c(1), t.d(1)), max(t.a(2), t.b(2), t.c(2), t.d(2));
				
					mesh[i] = t;
				}
				if(verbose_output)
					cout << "ElementCount: " << id << endl;
				cin >> s;
				assert(s == edele);
			}
		}
	}
	else
	{
		ifstream fin(fname.c_str());
		while(fin >> s)
		{
			if(s == nd)
			{
				fin >> input_count;
				vertex.resize(input_count + 1);
				for(int i = 1; i <= input_count; ++i)
				{
					fin >> id >> x_coord >> y_coord >> z_coord;
					assert(id == i);
					vertex[i] << x_coord, y_coord, z_coord;
				}

				if(verbose_output)
					cout << "NodeCount: " << id << endl;
				fin >> s;
				assert(s == ednd);
			}
			if(s == ele)
			{
				fin >> input_count;
				mesh.resize(input_count);
				for(int i = 1; i <= input_count; ++i)
				{
					fin >> id >> poly_face >> type >> id_A >> id_B >> id_C >> id_D;
					assert(id == i && poly_face == 4 && type == 0);
					tetrahedra t;
					t.a = vertex[id_A];
					t.b = vertex[id_B];
					t.c = vertex[id_C];
					t.d = vertex[id_D];
				
					t.bound_min << min(t.a(0), t.b(0), t.c(0), t.d(0)), min(t.a(1), t.b(1), t.c(1), t.d(1)), min(t.a(2), t.b(2), t.c(2), t.d(2));
					t.bound_max << max(t.a(0), t.b(0), t.c(0), t.d(0)), max(t.a(1), t.b(1), t.c(1), t.d(1)), max(t.a(2), t.b(2), t.c(2), t.d(2));
				
					mesh[i] = t;
				}
				if(verbose_output)
					cout << "ElementCount: " << id << endl;
				fin >> s;
				assert(s == edele);
			}
		}
	}
	return;
}

void NodesElementsInput(string fname)
{
	NodesInput((fname + ".node").c_str());
	ElementInput((fname + ".ele").c_str());
}

void StandardInput()
{
	int T;
	cin >> resolution;
	cin >> T;
	while(T--)
	{
		tetrahedra t;
		float inx, iny, inz;
		cin >> inx >> iny >> inz;
		t.a << inx, iny, inz;
		cin >> inx >> iny >> inz;
		t.b << inx, iny, inz;
		cin >> inx >> iny >> inz;
		t.c << inx, iny, inz;
		cin >> inx >> iny >> inz;
		t.d << inx, iny, inz;

		t.bound_min << min(t.a(0), t.b(0), t.c(0), t.d(0)), min(t.a(1), t.b(1), t.c(1), t.d(1)), min(t.a(2), t.b(2), t.c(2), t.d(2));
		t.bound_max << max(t.a(0), t.b(0), t.c(0), t.d(0)), max(t.a(1), t.b(1), t.c(1), t.d(1)), max(t.a(2), t.b(2), t.c(2), t.d(2));
		
		mesh.push_back(t);
	}
}

void BoundingBox()
{
	size_t mesh_size = mesh.size();
	#ifdef _OPENMP
	#pragma omp parallel for num_threads(8) if(parallel)
	#endif
	for(int i = 0; i < mesh_size; ++i)
	{
		min_corner << min(mesh[i].bound_min(0), min_corner(0)), min(mesh[i].bound_min(1), min_corner(1)), min(mesh[i].bound_min(2), min_corner(2));
		max_corner << max(mesh[i].bound_max(0), max_corner(0)), max(mesh[i].bound_max(1), max_corner(1)), max(mesh[i].bound_max(2), max_corner(2));
	}
	voxelsize = (max_corner - min_corner).maxCoeff() / resolution;
}

__device__
float deter(Matrix3f M)
{
	return M(0,0) * (M(1,1)*M(2,2) - M(1,2)*M(2,1)) - M(0,1) * (M(1,0)*M(2,2) - M(1,2)*M(2,0)) + M(0,2) * (M(1,0)*M(2,1) - M(1,1)*M(2,0));
}

__device__
bool Philipp(Vector3f A, Vector3f B, Vector3f C, Vector3f D, Vector3f P)
{
	Vector3f a = A - P, b = B - P, c = C - P, d = D - P;
	Matrix3f Da, Db, Dc, Dd;
	Da << b, c, d;
	Db << a, c, d;
	Dc << a, b, d;
	Dd << a, b, c;

	if((deter(Da) * deter(Dc) >= 0) && (deter(Db) * deter(Dd) >= 0) && (deter(Da) * deter(Db) <= 0))
		return true;
	return false;
}

__device__
int LocateVoxel(int reso, int tid)
{
	int yz = tid / reso;
	int z = yz / reso;
	int y = yz % reso;
	int x = tid % reso;
	return (z * MAXLENGTH + y) * MAXLENGTH + x;
}

__global__
void MeshIterKernel(int mesh_size, int resolution, bool *vox_d, tetrahedra* mesh_d, float *sample_coord_x, float *sample_coord_y, float *sample_coord_z, float voxelsize)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= mesh_size)return;

	Vector3f min_corner;
	min_corner << sample_coord_x[0], sample_coord_y[0], sample_coord_z[0];
	Vector3f start_sample = mesh_d[tid].bound_min - min_corner;
	Vector3f end_sample = mesh_d[tid].bound_max - min_corner;
	Vector3f st = start_sample / voxelsize;
	Vector3f ed = end_sample / voxelsize;

	if(tid == 0)
	{
		printf("Mesh_size: %d, Resolutoin: %d, Voxelsize: %f", mesh_size, resolution, voxelsize);
	}
	for(int i = int(st(0)); i <= ed(0); ++i)
	{
		for(int j = int(st(1)); j <= ed(1); ++j)
		{
			for(int k = int(st(2)); k <= ed(2); ++k)
			{
				if(vox_d[GetIndex(i,j,k)]) continue;
				Vector3f p;
				p << sample_coord_x[i], sample_coord_y[j], sample_coord_z[k];
				if( Philipp(mesh_d[tid].a, mesh_d[tid].b, mesh_d[tid].c, mesh_d[tid].d, p) )
				{
					vox_d[GetIndex(i,j,k)] = 1;
				}
			}
		}
	}
	return;
}

void voxelize()
{
	sample_coord_x.resize(resolution);
	sample_coord_y.resize(resolution);
	sample_coord_z.resize(resolution);
	sample_coord_x[0] = min_corner(0);
	sample_coord_y[0] = min_corner(1);
	sample_coord_z[0] = min_corner(2);
	// lookup table for the coords
	for(int i = 1; i < resolution; ++i)
	{
		sample_coord_x[i] = sample_coord_x[i - 1] + voxelsize;
		sample_coord_y[i] = sample_coord_y[i - 1] + voxelsize;
		sample_coord_z[i] = sample_coord_z[i - 1] + voxelsize;
	}

	bool *vox_d;
	tetrahedra* mesh_d;
	float *sample_coord_x_d, *sample_coord_y_d, *sample_coord_z_d;
	size_t mesh_size = mesh.size();
	size_t host_data_size = mesh_size * sizeof(tetrahedra);
	size_t sample_coord_size = resolution * sizeof(float);

	hipMalloc((void**)&vox_d, MAXSIZE * sizeof(bool));
	hipMalloc((void**)&mesh_d, host_data_size);
	hipMalloc((void**)&sample_coord_x_d, sample_coord_size);
	hipMalloc((void**)&sample_coord_y_d, sample_coord_size);
	hipMalloc((void**)&sample_coord_z_d, sample_coord_size);

	hipMemcpy(mesh_d, &mesh[0], host_data_size, hipMemcpyHostToDevice);
	hipMemcpy(sample_coord_x_d, &sample_coord_x[0], sample_coord_size, hipMemcpyHostToDevice);
	hipMemcpy(sample_coord_y_d, &sample_coord_y[0], sample_coord_size, hipMemcpyHostToDevice);
	hipMemcpy(sample_coord_z_d, &sample_coord_z[0], sample_coord_size, hipMemcpyHostToDevice);

	int grid = (mesh_size / BLOCK) + 1;

	MeshIterKernel<<<grid,BLOCK>>>(mesh_size, resolution, vox_d, mesh_d, sample_coord_x_d, sample_coord_y_d, sample_coord_z_d, voxelsize);
	hipDeviceSynchronize();

	hipMemcpy(voxels, vox_d, MAXSIZE * sizeof(bool), hipMemcpyDeviceToHost);

	hipFree(vox_d);
	hipFree(mesh_d);
	hipFree(sample_coord_x_d);
	hipFree(sample_coord_y_d);
	hipFree(sample_coord_z_d);
}

void VoxelOutput(string fname = "")
{
	if(fname.empty())
	{
		printf("%d x %d x %d\n", resolution, resolution, resolution);
		for(int i = 0; i < resolution; ++i)
		{
			for(int j = 0; j < resolution; ++j)
			{
				for(int k = 0; k < resolution; ++k)
				{
					cout << voxels[i][j][k] << " ";
				}
			}
		}
		cout << endl;
	}
	else
	{
		ofstream fout(fname.c_str());
		fout << "Resolution: " << resolution << "x" << resolution << "x" << resolution << endl;
		for(int i = 0; i < resolution; ++i)
		{
			for(int j = 0; j < resolution; ++j)
			{
				for(int k = 0; k < resolution; ++k)
				{
					fout << voxels[i][j][k] << " ";
				}
			}
		}
		fout << endl;
	}
}

int main(int argc, char** argv)
{
	cxxopts::Options options("exe.out", "One line description of MyProgram");
	options.add_options()
	("p,parallel", "Enable parallel computation on CUDA") // a bool parameter
	("n,nodes", "Enable nodes and elements input")
  ("r,resolution", "adjust the resolution of voxel output", cxxopts::value<int>())
  ("i,input", "Input file path", cxxopts::value<std::string>())
  ("o,output", "Output file path", cxxopts::value<std::string>())
  ("v,verbose", "Verbose output", cxxopts::value<bool>()->default_value("false"))
  ("h,help", "List all options")
  ;

  auto result = options.parse(argc, argv);
  
  if(result.count("help"))
  {
    cout << options.help({"", "Group"}) << endl;
    return 0;
  }

  if(result.count("parallel"))
  {
  	parallel = true;
  }

 	if(result.count("resolution"))
 	{
 		resolution = result["resolution"].as<int>();
 		cout << "Output resolution is set to " << resolution << "x" << resolution << "x" << resolution << endl;
 	}

 	if(result.count("verbose"))
 	{
 		verbose_output = result["verbose"].as<bool>();
 	}

	if(result.count("nodes"))
	{
		if(result.count("input"))
		{
			NodesElementsInput(result["input"].as<string>());
		}
		else
		{
			cout << "Nodes input requires specify input file path!\n";
			exit(0);
		}
	} 	
	else if(result.count("input"))
	{
		MshInput(result["input"].as<string>());
	}
	else
	{
		MshInput();

	}
	min_corner << INF, INF, INF;
	max_corner << -INF, -INF, -INF;
	BoundingBox();

	if(verbose_output)
		cout << min_corner << endl << endl << max_corner << endl;

  voxelize();

  if(result.count("output"))
  {
  	VoxelOutput(result["output"].as<string>());
  }
  else
  {
		VoxelOutput();
  }
	hipError_t err = hipGetLastError();  // add
	if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; // add
	hipProfilerStop();

	return 0;
}
